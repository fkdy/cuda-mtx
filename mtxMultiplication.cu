#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include"mtxMultiplication.h"
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include"assist.h"

// square matrix multiplication
// used for checking the results
void mtxMultiplication(int *m, int *n, int *p, int rows)
{
	int i, j, k, temp, isNULL=0;
	if(m==NULL || n==NULL){
		fprintf(stdout, "input matrix pointer is NULL\n");
		exit(-1);
	}else if(p==NULL){
		isNULL=1;
		p=(int *)malloc(rows*rows*sizeof(int));
	}
	for(i=0; i<rows; i++){
		for(j=0; j<rows; j++){
			p[i*rows + j]=0;
			for(k=0; k<rows; k++){
				temp=m[i*rows + k]*n[k*rows + j];
				p[i*rows + j]=p[i*rows + j]+temp;
			}
		}
	}
//	writeMtxFile("results.txt", p, rows, rows);
	if(isNULL)
		free(p);
}

__global__ void mtxMultKernel(int *pmDev, int *pnDev, int *ppDev, int tileRow, int mtxRow);
__global__ void mtxMultKernelGlobal(int *pmDev, int *pnDev, int *ppDev, int tileRow, int mtxRow);

// function port to cuda device
void mtxMultCUDA(int *pm, int *pn, int *pp, int rows)
{
	int size=rows*rows*sizeof(int);
	int *pmDev, *pnDev, *ppDev;
	// allocate memory on device
	hipMalloc((void**)&pmDev, size);
	hipMalloc((void**)&pnDev, size);
	hipMalloc((void**)&ppDev, size);
	// copy content of the matrix from host RAM to device global memory
	hipMemcpy(pmDev, pm, size, hipMemcpyHostToDevice);
	hipMemcpy(pnDev, pn, size, hipMemcpyHostToDevice);

	// kernel invocation code
	// 1.kernel configuration
	dim3 dimBlock(rows/4, rows/4);
	dim3 dimGrid(4,4);
	// 2.lunch the kernel
	mtxMultKernel<<<dimGrid, dimBlock>>>(pmDev, pnDev, ppDev, 4, rows);
//	mtxMultKernelGlobal<<<dimGrid, dimBlock>>>(pmDev, pnDev, ppDev, 4, rows);
	
	// copy content of the result matrix form device to host
	hipMemcpy(pp, ppDev, size, hipMemcpyDeviceToHost);
	// free allocated device memory
	hipFree(ppDev);
	hipFree(pmDev);
	hipFree(pnDev);
}

//1.matrix multipliction kernel function by using global memory access
__global__ void mtxMultKernelGlobal(int *pmDev, int *pnDev, int *ppDev, int tileRow, int mtxRow)
{
	// thread id 
	//   n
	// m p
	// tx corresponding to matrix column indice
	// tx -> matrix row number
	int c=blockIdx.x*tileRow+ threadIdx.x;
	int r=blockIdx.y*tileRow+ threadIdx.y;
	int i;
	ppDev[r*mtxRow + c]=0;
	for(i=0; i<mtxRow; i++)
		ppDev[r*mtxRow + c]+=pmDev[r*mtxRow + i]*pnDev[i*mtxRow + c];
}

//2 using shared memory
__global__ void mtxMultKernel(int *pmDev, int *pnDev, int *ppDev, int tileRow, int mtxRow)
{
	//   n
	// m p
	__shared__ int pmShared[4][16]; //[mtxRow/tileRow][mtxRow];
	__shared__ int pnShared[16][4]; //[mtxRow][mtxRow/tileRow];
	int c=blockIdx.x*tileRow+ threadIdx.x;
	int r=blockIdx.y*tileRow+ threadIdx.y;
	int temp, i;
	// each thread load mtxRow/tileRow byte of data into shared memory
	for(temp=0; temp < mtxRow/tileRow; temp++){
		pmShared[threadIdx.y][threadIdx.x*mtxRow/tileRow+temp]=pmDev[r*mtxRow + temp+threadIdx.x*mtxRow/tileRow];
		pnShared[threadIdx.y*mtxRow/tileRow+temp][threadIdx.x]=pnDev[mtxRow*(temp+threadIdx.y*mtxRow/tileRow) + c];
	}
	__syncthreads();
//	ppDev[r*mtxRow + c] = pmShared[threadIdx.y][c];
	// matrix multiplication
	temp=0;
	for(i=0; i<mtxRow; i++)
		temp+=pmShared[threadIdx.y][i]*pnShared[i][threadIdx.x];
	__syncthreads();
	ppDev[r*mtxRow + c]=temp;
}
