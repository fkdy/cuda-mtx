#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include"mtxMultiplication.h"
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include"assist.h"

// square matrix multiplication
void mtxMultiplication(int *m, int *n, int *p, int rows)
{
	int i, j, k, temp, isNULL=0;
	if(m==NULL || n==NULL){
		fprintf(stdout, "input matrix pointer is NULL\n");
		exit(-1);
	}else if(p==NULL){
		isNULL=1;
		p=(int *)malloc(rows*rows*sizeof(int));
	}
	for(i=0; i<rows; i++){
		for(j=0; j<rows; j++){
			p[i*rows + j]=0;
			for(k=0; k<rows; k++){
				temp=m[i*rows + k]*n[k*rows + j];
				p[i*rows + j]=p[i*rows + j]+temp;
			}
		}
	}
	writeMtxFile("results.txt", p, rows, rows);
	if(isNULL)
		free(p);
}

__global__ void mtxMultKernel(int *pmDev, int *pnDev, int *ppDev, int rows);

// function port to cuda device
void mtxMultCUDA(int *pm, int *pn, int *pp, int rows)
{
	int size=rows*rows*sizeof(int);
	int *pmDev, *pnDev, *ppDev;
	// allocate memory on device
	hipMalloc((void**)&pmDev, size);
	hipMalloc((void**)&pnDev, size);
	hipMalloc((void**)&ppDev, size);
	// copy content of the matrix from host to device
	hipMemcpy(pmDev, pm, size, hipMemcpyHostToDevice);
	hipMemcpy(pnDev, pn, size, hipMemcpyHostToDevice);

	// kernel invocation code
	// 1.kernel configuration
	dim3 dimBlock(rows, rows);
	dim3 dimGrid(1,1);
	// 2.lunch the kernel
	mtxMultKernel<<<dimGrid, dimBlock>>>(pmDev, pnDev, ppDev, rows);
	
	// copy content of the result matrix form device to host
	hipMemcpy(pp, ppDev, size, hipMemcpyDeviceToHost);
	// free allocated device memory
	hipFree(ppDev);
	hipFree(pmDev);
	hipFree(pnDev);

	// print out
	int i=0;
	for(i=0; i<rows*rows; i++){
		fprintf(stdout, "%d ", pp[i]);
	}
	fprintf(stdout, "\nthe matrix: ");
	for(i=0; i<rows*rows; i++){
		fprintf(stdout, "%d ", pm[i]);
	}
}

// matrix multipliction kernel function
__global__ void mtxMultKernel(int *pmDev, int *pnDev, int *ppDev, int rows)
{
	// thread id 
	//   n
	// m p
	// tx corresponding to matrix column indice
	// tx -> matrix row number
	int tx=threadIdx.x;
	int ty=threadIdx.y;
	int i=0;
	int temp=0;
	for(i=0; i<rows; i++)
		temp+=pmDev[ty*rows + i]*pnDev[i*rows + tx];
	ppDev[ty*rows + tx]=temp;
}
