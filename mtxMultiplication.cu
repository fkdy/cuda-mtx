#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include"mtxMultiplication.h"
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include"assist.h"

// square matrix multiplication
// used for checking the results
void mtxMultiplication(int *m, int *n, int *p, int rows)
{
	int i, j, k, temp, isNULL=0;
	if(m==NULL || n==NULL){
		fprintf(stdout, "input matrix pointer is NULL\n");
		exit(-1);
	}else if(p==NULL){
		isNULL=1;
		p=(int *)malloc(rows*rows*sizeof(int));
	}
	for(i=0; i<rows; i++){
		for(j=0; j<rows; j++){
			p[i*rows + j]=0;
			for(k=0; k<rows; k++){
				temp=m[i*rows + k]*n[k*rows + j];
				p[i*rows + j]=p[i*rows + j]+temp;
			}
		}
	}
	writeMtxFile("results.txt", p, rows, rows);
	if(isNULL)
		free(p);
}

__global__ void mtxMultKernel(int *pmDev, int *pnDev, int *ppDev, int tileRow, int mtxRow);

// function port to cuda device
void mtxMultCUDA(int *pm, int *pn, int *pp, int rows)
{
	int size=rows*rows*sizeof(int);
	int *pmDev, *pnDev, *ppDev;
	// allocate memory on device
	hipMalloc((void**)&pmDev, size);
	hipMalloc((void**)&pnDev, size);
	hipMalloc((void**)&ppDev, size);
	// copy content of the matrix from host to device
	hipMemcpy(pmDev, pm, size, hipMemcpyHostToDevice);
	hipMemcpy(pnDev, pn, size, hipMemcpyHostToDevice);

	// kernel invocation code
	// 1.kernel configuration
	dim3 dimBlock(rows/4, rows/4);
	dim3 dimGrid(4,4);
	// 2.lunch the kernel
	mtxMultKernel<<<dimGrid, dimBlock>>>(pmDev, pnDev, ppDev, 4, rows);
	
	// copy content of the result matrix form device to host
	hipMemcpy(pp, ppDev, size, hipMemcpyDeviceToHost);
	// free allocated device memory
	hipFree(ppDev);
	hipFree(pmDev);
	hipFree(pnDev);

	// print out
	int i=0;
	int j=0;
	for(i=0; i<rows; i++){
		for(j=0; j<rows; j++){
			fprintf(stdout, "%d ", pp[i*rows + j]);
		}
		fprintf(stdout, "\n");
	}
	fprintf(stdout, "\nthe matrix:\n");
	for(i=0; i<rows; i++){
		for(j=0; j<rows; j++){
			fprintf(stdout, "%d ", pm[i*rows + j]);
		}
		fprintf(stdout, "\n");
	}
/*	for(i=0; i<rows*rows; i++){
		fprintf(stdout, "%d ", pm[i]);
	}
*/
}

// matrix multipliction kernel function
__global__ void mtxMultKernel(int *pmDev, int *pnDev, int *ppDev, int tileRow, int mtxRow)
{
	// thread id 
	//   n
	// m p
	// tx corresponding to matrix column indice
	// tx -> matrix row number
	int c=blockIdx.x*tileRow+ threadIdx.x;
	int r=blockIdx.y*tileRow+ threadIdx.y;
	int i;
	ppDev[r*mtxRow + c]=0;
	for(i=0; i<mtxRow; i++)
		ppDev[r*mtxRow + c]+=pmDev[r*mtxRow + i]*pnDev[i*mtxRow + c];
}
